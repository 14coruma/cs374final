#include "hip/hip_runtime.h"
/* dsf-gpu-double.cu is a prime factorization program using Direct Search Factorization
 * and CUDA to parallelize and double precision.
 * This is good for factoring up to about 15 digits.
 *
 * Andrew Corum, Dec 2017
 * Usage: dsf [num] [num] [table format?]
 */

#include <stdio.h>
#include <omp.h>
#include <math.h>
#include <stdlib.h>
#include <gmp.h>
#include "cubi.h"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

__device__
void nextprime(cubi prime) {
	cubi one, two, prime_copy, R, Q;
	one        = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	two        = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	prime_copy = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	R          = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	Q          = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	cubi_init_d(prime_copy);
	cubi_init_d(one);
	cubi_init_d(two);
	cubi_init_d(R);
	cubi_init_d(Q);

	cubi_copy_d(prime, prime_copy);
	one[0] = 1;
	two[0] = 2;

	if(cubi_cmp_d(prime, two) < 0) {
		cubi_copy_d(two, prime);
	} else {
		cubi_div_d(prime, two, Q, R);
		if (cubi_cmp_d(R, one) != 0) {
			cubi_add_d(prime_copy, one, prime);
		} else {
			cubi_add_d(prime_copy, two, prime);
		}
	}

	free(one);
	free(two);
	free(prime_copy);
	free(R);
	free(Q);
}

/**
 * CUDA Kernel Device code
 */
__global__
void vectorHypot(
	cubi d_root, cubi d_N,
	cubi d_f1, cubi d_f2, cubi d_numThreads
)
{
	__shared__ int done;
	done = 0;

	// Creat cubis for all parameters
	cubi f1, f2, root, N, numThreads;
	root       = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	N          = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	f1         = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	f2         = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	numThreads = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	cubi_init_d(f1);
	cubi_init_d(f2);
	cubi_init_d(root);
	cubi_init_d(N);
	cubi_init_d(numThreads);

	for (int i = 0; i < SIZE; i++) {
		f1[i] = d_f1[i];
		f2[i] = d_f2[i];
		root[i] = d_root[i];
		N[i] = d_N[i];
	}
	numThreads[0] = d_numThreads[0];

	cubi id, prime, prime_copy, max, zero, R, Q;
	id         = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	prime      = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	prime_copy = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	max        = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	zero       = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	R          = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	Q          = (unsigned int*) malloc(SIZE * sizeof(unsigned int));
	cubi_init_d(id);
	cubi_init_d(prime);
	cubi_init_d(prime_copy);
	cubi_init_d(max);
	cubi_init_d(zero);
	cubi_init_d(R);
	cubi_init_d(Q);
	id[0] = blockDim.x * blockIdx.x + threadIdx.x + 1;

	// Figure out starting prime
	cubi_div_d(root, numThreads, prime, R);
	prime[0]++;
	cubi_mult_d(prime, id, max);
	id[0]--;
	cubi_copy_d(prime, prime_copy);
	cubi_mult_d(prime_copy, id, prime);
	nextprime(prime);
	nextprime(max);

	// Loop through potential factors
	while (cubi_cmp_d(prime, max) <= 0 && done == 0) {
		// If prime divides N, add prime and q to factors, then break
		cubi_div_d(N, prime, Q, R);
		if (cubi_cmp_d(R, zero) == 0) {
			cubi_copy_d(prime, f1);
			cubi_copy_d(Q, f2);
			done = 1;
			for (int i = 0; i < SIZE; i++) {
				d_f1[i] = f1[i];
				d_f2[i] = f2[i];
			}
			break;
		} else {
			// Otherwise figure out next prime and continue
			nextprime(prime);
		}
	}

	free(id);
	free(prime);
	free(prime_copy);
	free(max);
	free(Q);
	free(R);
	free(zero);
	free(f1);
	free(f2);
	free(N);
	free(root);
	free(numThreads);
}

void checkErr(hipError_t err, const char* msg) 
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{
	if (argc < 3) {
		fprintf(stderr, "ERROR: You need to provide to primes to multiply\n");
		exit(1);
	}

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

	size_t size = SIZE * sizeof(unsigned int);


	// Calculate root(N), since I don't have a cubi function for that yet
	unsigned long long LN1 = atoi(argv[1]);
	unsigned long long LN2 = atoi(argv[2]);
	unsigned long long LN = LN1 * LN2;
	char rootStr[SIZE * 6];
	sprintf(rootStr, "%.0f", ceil(sqrt(LN)));

	// Allocate the host input values
	cubi root, N, N1, N2;
	root = (unsigned int*) malloc(size);
	N    = (unsigned int*) malloc(size);
	N1   = (unsigned int*) malloc(size);
	N2   = (unsigned int*) malloc(size);
	cubi_init_h(root);
	cubi_init_h(N);
	cubi_init_h(N1);
	cubi_init_h(N2);

	cubi_set_str_bin_h(N1, argv[1]);
	cubi_set_str_bin_h(N2, argv[2]);
	cubi_mult_h(N1, N2, N);
	cubi_set_str_bin_h(root, rootStr);

	cubi h_f1         = (unsigned int*) malloc(size);
	cubi h_f2         = (unsigned int*) malloc(size);
	cubi h_N          = (unsigned int*) malloc(size);
	cubi h_root       = (unsigned int*) malloc(size);
	cubi h_numThreads = (unsigned int*) malloc(size);
    int threadsPerBlock = 256;
    int blocksPerGrid = 7;
    int numThreads;
	numThreads = threadsPerBlock * blocksPerGrid;
	numThreads = threadsPerBlock * blocksPerGrid;
	for (int i = 0; i < SIZE; i++) {
		h_N[i] = N[i];
		h_root[i] = root[i];
	}
	h_numThreads[0] = numThreads;

/*    // Verify that allocations succeeded
    if (h_f1 == NULL || h_f1 == NULL || root == NULL || N == NULL)
    {
        fprintf(stderr, "Failed to allocate host values!\n");
        exit(EXIT_FAILURE);
    }*/

    // 1a. Allocate the device input vectors A & B
/*    double *d_f1 = NULL;
    err = hipMalloc((void **)&d_f1, size);
    checkErr(err, "Failed to allocate device value d_f1");
    double *d_f2 = NULL;
    err = hipMalloc((void **)&d_f2, size);
    checkErr(err, "Failed to allocate device value d_f2");*/
	//cubi d_f1;
	//cubi_init_cuda(&d_f1, SIZE);
	//cubi d_f2;
	//cubi_init_cuda(&d_f2, SIZE);
	cubi d_f1 = NULL;
    err = hipMalloc((void **)&d_f1, size);
    checkErr(err, "Failed to allocate device value d_f1");
	cubi d_f2 = NULL;
    err = hipMalloc((void **)&d_f2, size);
    checkErr(err, "Failed to allocate device value d_f2");
	cubi d_root = NULL;
    err = hipMalloc((void **)&d_root, size);
    checkErr(err, "Failed to allocate device value d_root");
	cubi d_N = NULL;
    err = hipMalloc((void **)&d_N, size);
    checkErr(err, "Failed to allocate device value d_N");
	cubi d_numThreads = NULL;
    err = hipMalloc((void **)&d_numThreads, size);
    checkErr(err, "Failed to allocate device value d_numThreads");

	double wtime = -omp_get_wtime();
    // 2. Copy the host input vectors A and B in host memory 
    //     to the device input vectors in device memory
    //printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_f1, h_f1, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device value d_f1 from host to device");

    err = hipMemcpy(d_f2, h_f2, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device value d_f2 from host to device");
    
	err = hipMemcpy(d_root, h_root, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device value d_f1 from host to device");

    err = hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device value d_f2 from host to device");

    err = hipMemcpy(d_numThreads, h_numThreads, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device value d_f2 from host to device");

    // 3. Launch the Vector Add CUDA Kernel
	vectorHypot<<<blocksPerGrid, threadsPerBlock>>>(d_root, d_N, d_f1, d_f2, d_numThreads);
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorHypot kernel");

    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    //printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_f1, d_f1, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy value h_f1 from device to host");
    err = hipMemcpy(h_f2, d_f2, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy value h_f2 from device to host");
	wtime += omp_get_wtime();

/*    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrt(h_A[i] * h_A[i] + h_B[i] * h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }*/
	//printf("(%lu x %lu)\t(%lu x %lu)\n", N1, N2, *h_f1, *h_f2);
    //printf("CUDA test PASSED\n");

	cubi f1, f2;
	f1 = (unsigned int*) malloc(size);
	f2 = (unsigned int*) malloc(size);
	cubi_init_h(f1);
	cubi_init_h(f2);
	for (int i = 0; i < SIZE; i++) {
		f1[i] = h_f1[i];
		f2[i] = h_f2[i];
	}
	printf("base 10?\t%f\t(%s x %s)\n", /*ceil(log10(N)),*/ wtime,
		cubi_get_str_bin_h(f1), cubi_get_str_bin_h(f2));

    // Free device global memory
    err = hipFree(d_f1);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_f2);
    checkErr(err, "Failed to free device vector B");
	//cubi_free_cuda(&d_f1);
	//cubi_free_cuda(&d_f2);

    // Free host memory
    //cubi_free_h(&h_f1);
    //cubi_free_h(&h_f2);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    return 0;
}

